#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void dotProduct( float* matrixA, float* matrixB, float* matrixC, int n) {
 
   int j = blockIdx.x * blockDim.x + threadIdx.x;
   int i = blockIdx.y * blockDim.y + threadIdx.y;
      
   float value = 0;
   int k;
   for (k = 0; k < n; k++) {  	    	 
      value += matrixA[i * n + k] * matrixB[k * n + j];           
   }  

   matrixC[i * n + j] = value;
}


int main() {
	
	int size = 20;
	//host
	float *matrixA = (float*) malloc(size * size * sizeof(float));
	float *matrixB = (float*) malloc(size * size * sizeof(float));
	float *matrixC = (float*) malloc(size * size * sizeof(float));
	//device
	float *matrixA_d;
	float *matrixB_d;
	float *matrixC_d;

	int i = 0;

	while (i < size * size) {
		matrixA[i] = rand()%100;
		matrixB[i] = 1;		
		i++;
	}	

	
	
	hipMalloc((void**)&matrixA_d, size * size * sizeof(float)); 
	hipMalloc((void**)&matrixB_d, size * size * sizeof(float));
	hipMalloc((void**)&matrixC_d, size * size * sizeof(float)); 

	dim3 dimBlock(size , size);
   	dim3 dimGrid(size / dimBlock.x , size / dimBlock.y);

	hipMemcpy(matrixA_d, matrixA, size * size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(matrixB_d, matrixB, size * size * sizeof(float), hipMemcpyHostToDevice);



	dotProduct<<<dimGrid,dimBlock>>>(matrixA_d,matrixB_d,matrixC_d,size);	
	

	hipMemcpy(matrixC, matrixC_d, size * size * sizeof(float), hipMemcpyDeviceToHost);
	

	for (i = 0; i < size * size; i ++) {
		if(i > 0 && i % size == 0)
			printf("\n");
		printf("%.2f ", matrixC[i]);
	}

	printf("\n");
}