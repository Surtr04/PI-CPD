#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication and is exactly the same as
 * Chapter 7 of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * CUBLAS provides high-performance matrix multiplication.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include <matrixMul_kernel.cu>
#include <matrixMul_naive.cu>
#include <matrixMul_tiling.cu>
#include <matrixMul_coalescing.cu>
#include <matrixMul_noBankConflict.cu>
#include <matrixMul_outerProduct.cu>
#include <matrixMul_fpPortion.cu>
#include <matrixMul_prefetch.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    runTest(argc, argv);

    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{

    /****************************************************/
    /*  Preparations                                    */
    /****************************************************/

    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
    float flop = 2 * (float)WC * (float)HC * (float)WA;

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
    
    // allocate device memory
    float* d_A;
    cutilSafeCall(hipMalloc((void**) &d_A, mem_size_A));
    float* d_B;
    cutilSafeCall(hipMalloc((void**) &d_B, mem_size_B));

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    cutilSafeCall(hipMalloc((void**) &d_C, mem_size_C));

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);

#if CHECK_RESULT == 1
    // compute reference solution
    float* reference = (float*) malloc(mem_size_C);
    computeGold(reference, h_A, h_B, HA, WA, WB);
    CUTBoolean res;
#endif

    unsigned int timer = 0;
    dim3 threads,grid;

    /****************************************************/
    /*  CUDA SDK example                                */
    /****************************************************/

    // create and start timer
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    // execute the kernel
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");
    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );
    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("SDK Sample\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", cutGetTimerValue(timer), flop/cutGetTimerValue(timer) / 1e+6);
    cutilCheckError(cutDeleteTimer(timer));
#if CHECK_RESULT == 1
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  naive implementation on GPU                     */
    /****************************************************/

#if ENABLE_NAIVE == 1

    // create and start timer
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );
    // naive implementation
    matrixMul_naive<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");
    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );
    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Naive\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", cutGetTimerValue(timer), flop/cutGetTimerValue(timer) / 1e+6);
    cutilCheckError(cutDeleteTimer(timer));
#if CHECK_RESULT == 1
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, h_C, WC, HC);
#endif

#endif
    /****************************************************/
    /*  Tiling without global mem coalescing            */
    /****************************************************/

    // create and start timer
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );
    // naive implementation
    matrixMul_tiling<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");
    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );
    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Tiling\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", cutGetTimerValue(timer), flop/cutGetTimerValue(timer) / 1e+6);
    cutilCheckError(cutDeleteTimer(timer));
#if CHECK_RESULT == 1
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Global mem coalescing with smem bank conflict   */
    /****************************************************/

    // create and start timer
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );
    // naive implementation
    matrixMul_coalescing<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");
    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );
    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Global mem coalescing\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", cutGetTimerValue(timer), flop/cutGetTimerValue(timer) / 1e+6);
    cutilCheckError(cutDeleteTimer(timer));
#if CHECK_RESULT == 1
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Global mem coalescing w/o smem bank conflict    */
    /****************************************************/

    // create and start timer
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );
    // naive implementation
    matrixMul_noBankConflict<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");
    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );
    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Remove shared mem bank conflict\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", cutGetTimerValue(timer), flop/cutGetTimerValue(timer) / 1e+6);
    cutilCheckError(cutDeleteTimer(timer));
#if CHECK_RESULT == 1
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Threads perform outer product                   */
    /****************************************************/

    // create and start timer
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, 4);
    grid = dim3(WC / (BLOCK_SIZE*4), HC / BLOCK_SIZE);
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );
    // naive implementation
    matrixMul_outerProduct<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");
    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );
    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Threads perform outer product\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", cutGetTimerValue(timer), flop/cutGetTimerValue(timer) / 1e+6);
    cutilCheckError(cutDeleteTimer(timer));
#if CHECK_RESULT == 1
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, h_C, WC, HC);
#endif


    /****************************************************/
    /*  Increasing fp Proportion                        */
    /****************************************************/

    // create and start timer
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, 4);
    grid = dim3(WC / (BLOCK_SIZE*4), HC / BLOCK_SIZE);
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );
    // naive implementation
    matrixMul_fpPortion<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");
    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );
    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Increase fp Portion\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", cutGetTimerValue(timer), flop/cutGetTimerValue(timer) / 1e+6);
    cutilCheckError(cutDeleteTimer(timer));
#if CHECK_RESULT == 1
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Prefetching                                     */
    /****************************************************/

    // create and start timer
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, 4);
    grid = dim3(WC / (BLOCK_SIZE*4), HC / BLOCK_SIZE);
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );
    // naive implementation
    matrixMul_prefetch<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");
    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );
    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Prefetching\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", cutGetTimerValue(timer), flop/cutGetTimerValue(timer) / 1e+6);
    cutilCheckError(cutDeleteTimer(timer));
#if CHECK_RESULT == 1
    // check result
    res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Cleaning                                        */
    /****************************************************/

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
#if CHECK_RESULT == 1
    free(reference);
#endif
    cutilSafeCall(hipFree(d_A));
    cutilSafeCall(hipFree(d_B));
    cutilSafeCall(hipFree(d_C));

    hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (fabs(data1[k] - data2[k]) > 0.1 ) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f \n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf(" nTotal Errors = %d n", error_count);
}

